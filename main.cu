#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define BLOCKY 8
#define BLOCKX 8
extern "C" {
    #include "libs/bitmap.h"
}

#define ERROR_EXIT -1

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5
// If you apply another filter, remember not only to exchange
// the filter but also the filterFactor and the correct dimension.

int const sobelYFilter[] = {-1, -2, -1,
                             0,  0,  0,
                             1,  2,  1};
float const sobelYFilterFactor = (float) 1.0;

int const sobelXFilter[] = {-1, -0, -1,
                            -2,  0, -2,
                            -1,  0, -1 , 0};
float const sobelXFilterFactor = (float) 1.0;


int const laplacian1Filter[] = {  -1,  -4,  -1,
                                 -4,  20,  -4,
                                 -1,  -4,  -1};
int const laplacian1filterDim=3;
float const laplacian1FilterFactor = (float) 1.0;

int const laplacian2Filter[] = { 0,  1,  0,
                                 1, -4,  1,
                                 0,  1,  0};
float const laplacian2FilterFactor = (float) 1.0;

int const laplacian3Filter[] = { -1,  -1,  -1,
                                  -1,   8,  -1,
                                  -1,  -1,  -1};
float const laplacian3FilterFactor = (float) 1.0;


//Bonus Filter:

int const gaussianFilter[] = { 1,  4,  6,  4, 1,
                               4, 16, 24, 16, 4,
                               6, 24, 36, 24, 6,
                               4, 16, 24, 16, 4,
                               1,  4,  6,  4, 1 };

float const gaussianFilterFactor = (float) 1.0 / 256.0;

#define PIXEL(i,j) ((i)+(j)*XSIZE)
// Apply convolutional filter on image data
__global__ void applyFilter(unsigned char *in, unsigned char *out, unsigned int XSIZE, unsigned int YSIZE, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);

  __shared__ unsigned char st[BLOCKX*BLOCKY];
  __shared__ int kerneld[laplacian1filterDim*laplacian1filterDim];

  int i = blockIdx.x*BLOCKX+threadIdx.x;
  int j = blockIdx.y*BLOCKY + threadIdx.y;


  if(i>0&&i<XSIZE&&j>0&&j<YSIZE)
  {
        int threadsum=threadIdx.x+threadIdx.y*BLOCKX;
        if(threadsum<filterDim*filterDim)
        {
          kerneld[threadsum]=filter[threadsum];
        }
        __syncthreads();
        st[threadsum]=in[PIXEL(i,j)];
        __syncthreads();


     int aggregate =0;
     for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = threadIdx.y + (ky - filterCenter);
          int xx = threadIdx.x + (kx - filterCenter);
          int mod= kerneld[nky * filterDim + nkx];
          if (xx >= 0 && xx < BLOCKX && yy >=0 && yy < BLOCKY)
          {
            aggregate += st[xx+yy*BLOCKX] * mod;
          }
          else
          {
            yy = j + (ky - filterCenter);
            xx = i + (kx - filterCenter);
            aggregate += in[PIXEL(xx,yy)] * mod;
          }

        }

      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[PIXEL(i,j)] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[PIXEL(i,j)] = 0;
      }

  }

}

__global__ void applyFilterNormal(unsigned char *in, unsigned char *out, unsigned int XSIZE, unsigned int YSIZE, int *filter, unsigned int filterDim, float filterFactor) {
  unsigned int const filterCenter = (filterDim / 2);

  int i = blockIdx.x*BLOCKX+threadIdx.x;
  int j = blockIdx.y*BLOCKY + threadIdx.y;

  if(i>0&&i<XSIZE&&j>0&&j<YSIZE)
  {
     int aggregate =0;

     for (unsigned int ky = 0; ky < filterDim; ky++) {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++) {
          int nkx = filterDim - 1 - kx;

          int yy = j + (ky - filterCenter);
          int xx = i + (kx - filterCenter);
          aggregate += in[PIXEL(xx,yy)] * filter[nky * filterDim + nkx];

        }

      }
      aggregate *= filterFactor;
      if (aggregate > 0) {
        out[PIXEL(i,j)] = (aggregate > 255) ? 255 : aggregate;
      } else {
        out[PIXEL(i,j)] = 0;
      }
  }

}






void help(char const *exec, char const opt, char const *optarg) {
    FILE *out = stdout;
    if (opt != 0) {
        out = stderr;
        if (optarg) {
            fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
        } else {
            fprintf(out, "Invalid parameter - %c\n", opt);
        }
    }
    fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
    fprintf(out, "\n");
    fprintf(out, "Options:\n");
    fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

    fprintf(out, "\n");
    fprintf(out, "Example: %s in.bmp out.bmp -i 10000\n", exec);
}

int main(int argc, char **argv) {
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  int ret = 0;

  static struct option const long_options[] =  {
      {"help",       no_argument,       0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}
  };

  static char const * short_options = "hi:";
  {
    char *endptr;
    int c;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1) {
      switch (c) {
      case 'h':
        help(argv[0],0, NULL);
        return 0;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg) {
          help(argv[0], c, optarg);
          return ERROR_EXIT;
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind+1)) {
    help(argv[0],' ',"Not enough arugments");
    return ERROR_EXIT;
  }
  input = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(input, argv[optind], strlen(argv[optind]));
  optind++;

  output = (char *)calloc(strlen(argv[optind]) + 1, sizeof(char));
  strncpy(output, argv[optind], strlen(argv[optind]));
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0,0);
  if (image == NULL) {
    fprintf(stderr, "Could not allocate new image!\n");
  }

  if (loadBmpImage(image, input) != 0) {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    return ERROR_EXIT;
  }


  // Create a single color channel image. It is easier to work just with one color
  bmpImageChannel *imageChannel = newBmpImageChannel(image->width, image->height);
  if (imageChannel == NULL) {
    fprintf(stderr, "Could not allocate new image channel!\n");
    freeBmpImage(image);
    return ERROR_EXIT;
  }

  // Extract from the loaded image an average over all colors - nothing else than
  // a black and white representation
  // extractImageChannel and mapImageChannel need the images to be in the exact
  // same dimensions!
  // Other prepared extraction functions are extractRed, extractGreen, extractBlue
  if(extractImageChannel(imageChannel, image, extractAverage) != 0) {
    fprintf(stderr, "Could not extract image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }


  //Here we do the actual computation!
  // imageChannel->data is a 2-dimensional array of unsigned char which is accessed row first ([y][x])
  int XSIZE=imageChannel->width;
  int YSIZE=imageChannel->height;
  unsigned char *cudachannel1;
  unsigned char *cudachannel2;
  int *kernel;

  int size = sizeof(unsigned char)*XSIZE*YSIZE;

  hipMalloc(&cudachannel1, size);
  hipMalloc(&cudachannel2, size);
  hipMalloc(&kernel, sizeof(laplacian1Filter));


  int iXSIZE=XSIZE;
  int iYSIZE=YSIZE;
  while(iXSIZE%BLOCKX!=0)
  {
     iXSIZE++;
  }
  while(iYSIZE%BLOCKY!=0)
  {
     iYSIZE++;
  }

  dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
  dim3 threadBlock(BLOCKX, BLOCKY);

  unsigned char *deviceMem = (unsigned char*)malloc(size);
  for(int x=0;x<XSIZE;x++)
  {
    for(int y=0;y<YSIZE;y++)
    {
      deviceMem[y*XSIZE+x]=imageChannel->data[y][x];
    }
  }
  hipMemcpy(kernel, laplacian1Filter, sizeof(laplacian1Filter),hipMemcpyHostToDevice);

  for (unsigned int i = 0; i < iterations; i ++) {
    hipMemcpy(cudachannel1,deviceMem, size,hipMemcpyHostToDevice);
    struct timespec start, end;
    clock_gettime(CLOCK_REALTIME, &start);
    applyFilter<<<gridBlock,threadBlock>>>(cudachannel1,cudachannel2,XSIZE,YSIZE,kernel,3, laplacian1FilterFactor);
    clock_gettime(CLOCK_REALTIME, &end);

    if (end.tv_nsec < start.tv_nsec) {
              end.tv_nsec += 1000000000;
              end.tv_sec--;
          }

          printf("%ld.%09ld  GPU time\n", (long)(end.tv_sec - start.tv_sec),
              end.tv_nsec - start.tv_nsec);
    hipMemcpy(deviceMem, cudachannel2, size,hipMemcpyDeviceToHost);

  }
  hipFree(kernel);
  hipFree(cudachannel1);
  hipFree(cudachannel2);

  for(int x=0;x<XSIZE;x++)
  {
    for(int y=0;y<YSIZE;y++)
    {
      imageChannel->data[y][x]=deviceMem[y*XSIZE+x];
    }
  }
  free(deviceMem);
  // Map our single color image back to a normal BMP image with 3 color channels
  // mapEqual puts the color value on all three channels the same way
  // other mapping functions are mapRed, mapGreen, mapBlue


  if (mapImageChannel(image, imageChannel, mapEqual) != 0) {
    fprintf(stderr, "Could not map image channel!\n");
    freeBmpImage(image);
    freeBmpImageChannel(imageChannel);
    return ERROR_EXIT;
  }
  freeBmpImageChannel(imageChannel);

  //Write the image back to disk
  if (saveBmpImage(image, output) != 0) {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    return ERROR_EXIT;
  };

  ret = 0;
  if (input)
    free(input);
  if (output)
    free(output);
  return ret;
};
